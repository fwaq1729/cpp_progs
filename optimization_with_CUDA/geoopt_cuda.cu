#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <string>
#include <vector>
#include <algorithm>
#include <stdexcept>

#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipblas.h>

#include "geoopt_cuda.h"

using namespace std;

// Error checking macro for CUDA calls
#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA Error at " << __FILE__ << ":" << __LINE__ << ": " << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

// Error checking macro for cuSOLVER calls
#define CUSOLVER_CHECK(call) \
    do { \
        hipsolverStatus_t status = call; \
        if (status != HIPSOLVER_STATUS_SUCCESS) { \
            std::cerr << "cuSOLVER Error at " << __FILE__ << ":" << __LINE__ << ": " << status << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

void geoopt_routines_with_cuda::fill_mat(vector<double>& A, const int nrow, const int ncol)
{
  for (int i = 0, count = 0; i != nrow; ++i)
  {
    for (int j = 0; j != ncol; ++count, ++j)
    {
       A[count] = rand() / static_cast<double>(RAND_MAX);
    }
  }
}

vector<double> geoopt_routines_with_cuda::get_colwise_matrix(const vector<double>& in, const int nrow, const int ncol)
{
  vector<double> ou(nrow * ncol);
  for (int i = 0, count = 0; i != nrow; ++i)
  {
    for (int j = 0; j != ncol; ++count, ++j)
    {
      ou[count] = in[i + nrow * j];
    }
  }

  return ou;
}

void geoopt_routines_with_cuda::prn_mat(const string tag, const vector<double>& A, const int nrow, const int ncol)
{
  cout << tag << " = np.array([\n";
  for (int i = 0; i != nrow; ++i)
  {
    cout << "  [";
    for (int j = 0; j != ncol; ++j)
    {
       if (j != ncol - 1)
       {
         cout << setw(15) << setprecision(8) << scientific << A[j + ncol * i] << ",";
       } else if (i != nrow - 1)
       {
         cout << setw(15) << setprecision(8) << scientific << A[j + ncol * i] << "],\n";
       } else
       {
         cout << setw(15) << setprecision(8) << scientific << A[j + ncol * i] << "]]);\n";
       }
    }
  }
}

void geoopt_routines_with_cuda::Geoopt_cuda::matmul_cublas(
  const int m,
  const int k,
  const int n,
  const double* h_A, const double* h_B, double* h_C)
{
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  double *d_A, *d_B, *d_C;
  hipMalloc(&d_A, m * k * sizeof(double));
  hipMalloc(&d_B, k * n * sizeof(double));
  hipMalloc(&d_C, m * n * sizeof(double));
  hipMemcpy(d_A, h_A, m * k * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, k * n * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_C, h_C, m * n * sizeof(double), hipMemcpyHostToDevice);
  // Perform matrix multiplication C = alpha * A * B + beta * C
  double alpha = 1.0;
  double beta = 0.0;
  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
              n, m, k, &alpha, d_B, n, d_A, k, &beta, d_C, n);
  hipMemcpy(h_C, d_C, m * n * sizeof(double), hipMemcpyDeviceToHost);
  hipblasDestroy(handle);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}

vector<double> geoopt_routines_with_cuda::Geoopt_cuda::linear_solver(const vector<double>& h_A, const vector<double>& h_B)
{
  const int n = h_B.size();
  vector<double> h_X(n);
  // Device-side pointers
  double *d_A, *d_B;
  int *d_P;
  double *d_Workspace; // Workspace for cuSOLVER routines
  int *d_Info;    // For error codes from cuSOLVER

  // Allocate device memory
  CUDA_CHECK(hipMalloc(&d_A, n * n * sizeof(double)));
  CUDA_CHECK(hipMalloc(&d_B, n * sizeof(double)));
  CUDA_CHECK(hipMalloc(&d_P, n * sizeof(int)));
  CUDA_CHECK(hipMalloc(&d_Info, sizeof(int)));

  // Copy data from host to device
  CUDA_CHECK(hipMemcpy(d_A, h_A.data(), n * n * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_B, h_B.data(), n * sizeof(double), hipMemcpyHostToDevice));

  // Initialize cuSOLVER handle
  hipsolverHandle_t cusolverH = NULL;
  CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));

  // Query workspace size for LU factorization
  int lwork = 0;
  CUSOLVER_CHECK(hipsolverDnDgetrf_bufferSize(cusolverH, n, n, d_A, n, &lwork));
  CUDA_CHECK(hipMalloc(&d_Workspace, lwork * sizeof(double)));

  // Perform LU factorization
  CUSOLVER_CHECK(hipsolverDnDgetrf(cusolverH, n, n, d_A, n, d_Workspace, d_P, d_Info));

  // Check for singularity (if *d_Info != 0, matrix is singular)
  int info;
  CUDA_CHECK(hipMemcpy(&info, d_Info, sizeof(int), hipMemcpyDeviceToHost));
  if (info != 0) {
      std::cerr << "Matrix A is singular. Cannot solve the system." << std::endl;
      // Handle singularity appropriately (e.g., return, use a different solver)
  } else {
      // Solve the linear system A*X = B using the LU factors
      CUSOLVER_CHECK(hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, n, 1, d_A, n, d_P, d_B, n, d_Info));
      // Copy solution from device to host
      CUDA_CHECK(hipMemcpy(h_X.data(), d_B, n * sizeof(double), hipMemcpyDeviceToHost));
    }
  // Clean up
  CUDA_CHECK(hipFree(d_A));
  CUDA_CHECK(hipFree(d_B));
  CUDA_CHECK(hipFree(d_P));
  CUDA_CHECK(hipFree(d_Workspace));
  CUDA_CHECK(hipFree(d_Info));
  CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));

  return h_X;
}

tuple<vector<double>, vector<double>> geoopt_routines_with_cuda::Geoopt_cuda::eigenvalue_solver(
  const int ndata,
  const vector<double>& h_A)
{
    const int ndata2 = ndata * ndata;
    // Pointers for device memory
    double *d_A, *d_W, *d_work;
    int *d_info;

    // cuSOLVER variables
    hipsolverHandle_t cusolverH = NULL;
    int lwork = 0;

    // 1. Initialize cuSOLVER handle
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));

    // 2. Allocate device memory
    CUDA_CHECK(hipMalloc((void**)&d_A, ndata2 * sizeof(double)));
    CUDA_CHECK(hipMalloc((void**)&d_W, ndata * sizeof(double)));
    CUDA_CHECK(hipMalloc((void**)&d_info, sizeof(int)));

    // 3. Copy host matrix to device
    CUDA_CHECK(hipMemcpy(d_A, h_A.data(), ndata2 * sizeof(double), hipMemcpyHostToDevice));

    // 4.1. Query for the optimal work buffer size
    CUSOLVER_CHECK(hipsolverDnDsyevd_bufferSize(
        cusolverH,
        HIPSOLVER_EIG_MODE_VECTOR, // HIPSOLVER_EIG_MODE_VECTOR for both eigenvalues and eigenvectors
        HIPBLAS_FILL_MODE_LOWER,   // Lower triangular part of the matrix
        ndata,                    // Matrix dimension
        d_A,                      // Device pointer to the matrix
        ndata,                    // Leading dimension of d_A
        d_W,                      // Device pointer for eigenvalues
        &lwork                    // Pointer to workspace size
    ));

    // 4.2. Allocate device work buffer
    CUDA_CHECK(hipMalloc((void**)&d_work, lwork * sizeof(double)));

    // 4.3. Compute eigenvalues and eigenvectors
    // The eigenvectors are stored in the input matrix d_A
    CUSOLVER_CHECK(hipsolverDnDsyevd(
        cusolverH,
        HIPSOLVER_EIG_MODE_VECTOR,
        HIPBLAS_FILL_MODE_LOWER,
        ndata,
        d_A,
        ndata,
        d_W,
        d_work,
        lwork,
        d_info
    ));

    // 5. Copy results back to host
    std::vector<double> h_eigenvalues(ndata);
    std::vector<double> h_eigenvectors(ndata2);

    CUDA_CHECK(hipMemcpy(h_eigenvalues.data(), d_W, ndata * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_eigenvectors.data(), d_A, ndata2 * sizeof(double), hipMemcpyDeviceToHost));

    int h_info;
    CUDA_CHECK(hipMemcpy(&h_info, d_info, sizeof(int), hipMemcpyDeviceToHost));

    if (h_info != 0) {
        const string tag = "cuSOLVER failed to converge. Info = " + to_string(h_info);
        throw runtime_error(tag);
    }

    // 6. Clean up
    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_W));
    CUDA_CHECK(hipFree(d_work));
    CUDA_CHECK(hipFree(d_info));

    return make_tuple(h_eigenvalues, h_eigenvectors);
}

void geoopt_routines_with_cuda::Geoopt_cuda::daxpy_cublas(
  vector<double>& h_y,
  const vector<double>& h_x,
  const double alpha)
{
    // Device-side pointers
    double *d_x, *d_y;

    const int n = h_y.size();
    // Allocate memory on the device
    hipMalloc(&d_x, n * sizeof(double));
    hipMalloc(&d_y, n * sizeof(double));

    // Copy host data to device
    hipMemcpy(d_x, h_x.data(), n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y.data(), n * sizeof(double), hipMemcpyHostToDevice);

    // Initialize cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Perform the AXPY operation: d_y = alpha * d_x + d_y
    // Parameters: handle, n, alpha, d_x, incx, d_y, incy
    // incx and incy are strides (1 for contiguous elements)
    hipblasDaxpy(handle, n, &alpha, d_x, 1, d_y, 1);

    // Copy results back to host
    hipMemcpy(h_y.data(), d_y, n * sizeof(double), hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_x);
    hipFree(d_y);
    hipblasDestroy(handle);
}

// Computing: proj = dot(B, B_inv)
vector<double> geoopt_routines_with_cuda::Geoopt_cuda::get_proj(
  const int ndata,
  const int nrow_B,
  const int ncol_B,
  const int ncol_Binv,
  const vector<double>& B,     // Wilson matrix
  const vector<double>& B_inv) // inverse of Wilson matrix
{
  vector<double> proj(ndata * ndata);
  matmul_cublas(nrow_B, ncol_B, ncol_Binv, B.data(), B_inv.data(), proj.data());

  return proj;
}

// Computing: g_new = dot(proj, s.interpolated.g)
vector<double> geoopt_routines_with_cuda::Geoopt_cuda::get_g_new(
  const vector<double>& proj,
  const vector<double>& g_interp)
{
  const int ndata = g_interp.size();
  vector<double> g_new(ndata);
  matmul_cublas(ndata, ndata, 1, proj.data(), g_interp.data(), g_new.data());

  return g_new;
}

// Computing: H_proj = proj.dot(s.H).dot(proj) + 1000 * (eye(len(s.coords)) - proj)
vector<double> geoopt_routines_with_cuda::Geoopt_cuda::get_Hproj(
  const int ndata,
  const vector<double>& proj,
  const vector<double>& H)
{
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  double *d_A, *d_B, *d_C;
  const int ndata2 = ndata * ndata;
  hipMalloc(&d_A, ndata2 * sizeof(double));
  hipMalloc(&d_B, ndata2 * sizeof(double));
  hipMalloc(&d_C, ndata2 * sizeof(double));
  hipMemcpy(d_A, proj.data(), ndata2 * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_B, H.data(), ndata2 * sizeof(double), hipMemcpyHostToDevice);
  double alpha = 1.0;
  double beta = 0.0;
  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
              ndata, ndata, ndata, &alpha, d_B, ndata, d_A, ndata, &beta, d_C, ndata);
  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
              ndata, ndata, ndata, &alpha, d_A, ndata, d_C, ndata, &beta, d_C, ndata);
  double alpha1 = -1000.0;
  hipblasDaxpy(handle, ndata2, &alpha1, d_A, 1, d_C, 1);
  vector<double> Hproj(ndata2);
  hipMemcpy(Hproj.data(), d_C, ndata2 * sizeof(double), hipMemcpyDeviceToHost);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  hipblasDestroy(handle);
  for (int i = 0; i != ndata; ++i)
  {
    Hproj[i + ndata * i] -= alpha1;
  }

  return Hproj;
}

// Computing:
// dq, dE, on_sphere = quadratic_step(dot(proj, s.interpolated.g), H_proj, s.weights, s.trust, log=log)
tuple<double, vector<double>, bool> geoopt_routines_with_cuda::Geoopt_cuda::do_quadratic_step(
  const double trust,
  const vector<double>& g_new,
  const vector<double>& Hproj)
{
  const int ndata = g_new.size();
  const int ndata1 = ndata + 1;
  vector<double> rfo_symm(ndata1 * ndata1);
  for (int j = 0; j != ndata; ++j)
  {
    rfo_symm[ndata1 - 1 + ndata1 * j] = g_new[j];
    rfo_symm[j + ndata1 * (ndata1 - 1)] = g_new[j];
    rfo_symm[j + ndata1 * j] = Hproj[j + ndata * j];
    for (int i = j + 1; i != ndata; ++i)
    {
      rfo_symm[i + ndata1 * j] = 0.5 * (Hproj[i + ndata * j] + Hproj[j + ndata * i]);
      rfo_symm[j + ndata1 * i] = rfo_symm[i + ndata1 * j];
    }
  }
  vector<double> eigenvals1(ndata1);
  vector<double> eigenvectors1(ndata1 * ndata1);
  tie(eigenvals1, eigenvectors1) = eigenvalue_solver(ndata1, rfo_symm);

  vector<double> dq(ndata);
  for (int j = 0; j != ndata; j++)
  {
    dq[j] = eigenvectors1[j + ndata1 * 0] / eigenvectors1[ndata + ndata1 * 0];
  }
  vector<double> Hdq(ndata);
  matmul_cublas(ndata, ndata, 1, Hproj.data(), dq.data(), Hdq.data());
  double dE1, dE2, norm_dq;
  double *g_new_d, *dq_d, *Hdq_d;
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipMalloc(&g_new_d, ndata * sizeof(double));
  hipMalloc(&dq_d, ndata * sizeof(double));
  hipMalloc(&Hdq_d, ndata * sizeof(double));
  hipMemcpy(g_new_d, g_new.data(), ndata * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dq_d, dq.data(), ndata * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(Hdq_d, Hdq.data(), ndata * sizeof(double), hipMemcpyHostToDevice);
  hipblasDdot(handle, ndata, g_new_d, 1, dq_d, 1, &dE1);
  hipblasDdot(handle, ndata, Hdq_d, 1, dq_d, 1, &dE2);
  const double dE = dE1 + 0.5 * dE2;
  hipblasDdot(handle, ndata, dq.data(), 1, dq.data(), 1, &norm_dq);
  hipblasDestroy(handle);
  hipFree(g_new_d);
  hipFree(dq_d);
  hipFree(Hdq_d);
  norm_dq = sqrt(norm_dq);
  const bool on_sphere = (norm_dq <= trust) ? false : true;

  return make_tuple(dE, dq, on_sphere);
}

vector<double> geoopt_routines_with_cuda::Geoopt_cuda::update_Hessian_BFGS(
  const vector<double>& q,
  const vector<double>& best_q,
  const vector<double>& g,
  const vector<double>& best_g,
  const vector<double>& H)
{
  const int ndata = q.size();
  const int ndata2 = ndata * ndata;
  double *q_h, *g_h, *q_h1, *g_h1, *dH1, *dH2_0, *H_h, *HdH2_0, *HdH2_0H, *dqH;
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipMalloc(&q_h, ndata * sizeof(double));
  hipMalloc(&g_h, ndata * sizeof(double));
  hipMalloc(&q_h1, ndata * sizeof(double));
  hipMalloc(&g_h1, ndata * sizeof(double));
  hipMalloc(&dH1, ndata2 * sizeof(double));
  hipMalloc(&dH2_0, ndata2 * sizeof(double));
  hipMalloc(&H_h, ndata2 * sizeof(double));
  hipMalloc(&HdH2_0, ndata2 * sizeof(double));
  hipMalloc(&HdH2_0H, ndata2 * sizeof(double));
  hipMalloc(&dqH, ndata * sizeof(double));
  hipMemcpy(q_h, q.data(), ndata * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(g_h, g.data(), ndata * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(q_h1, best_q.data(), ndata * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(g_h1, best_g.data(), ndata * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(H_h, H.data(), ndata2 * sizeof(double), hipMemcpyHostToDevice);
  double alpha0 = -1.0;
  hipblasDaxpy(handle, ndata, &alpha0, q_h1, 1, q_h, 1);
  hipblasDaxpy(handle, ndata, &alpha0, g_h1, 1, g_h, 1);

  double alpha = 1.0;
  double beta = 0.0;
  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
              ndata, ndata, 1, &alpha, g_h, ndata, g_h, 1, &beta, dH1, ndata);
  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
              ndata, ndata, 1, &alpha, q_h, ndata, q_h, 1, &beta, dH2_0, ndata);
  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
              ndata, ndata, ndata, &alpha, dH2_0, ndata, H_h, ndata, &beta, HdH2_0, ndata);
  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
              ndata, ndata, ndata, &alpha, H_h, ndata, HdH2_0, ndata, &beta, HdH2_0H, ndata);
  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
              1, ndata, ndata, &alpha, q_h, 1, H_h, ndata, &beta, dqH, 1);

  double qd_dot, dqHdq_dot;
  hipblasDdot(handle, ndata, g_h, 1, q_h, 1, &qd_dot);
  hipblasDdot(handle, ndata, dqH, 1, q_h, 1, &dqHdq_dot);
  const double dqdg = 1.0 / qd_dot;
  const double dqHdq = 1.0 / dqHdq_dot;
  hipblasDscal(handle, ndata2, &dqdg, dH1, 1);
  hipblasDscal(handle, ndata2, &dqHdq, HdH2_0H, 1);
  double alpha1 = -1.0;
  double alpha2 =  1.0;
  hipblasDaxpy(handle, ndata2, &alpha1, HdH2_0H, 1, dH1, 1);
  hipblasDaxpy(handle, ndata2, &alpha2, dH1, 1, H_h, 1);
  vector<double> H_updated(ndata2);
  hipMemcpy(H_updated.data(), H_h, ndata2 * sizeof(double), hipMemcpyDeviceToHost);
  hipblasDestroy(handle);
  hipFree(q_h);
  hipFree(g_h);
  hipFree(q_h1);
  hipFree(g_h1);
  hipFree(dH1);
  hipFree(dH2_0);
  hipFree(H_h);
  hipFree(HdH2_0);
  hipFree(HdH2_0H);
  hipFree(dqH);

  return H_updated;
}
